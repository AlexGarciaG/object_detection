#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/core/cuda.hpp>

__global__ void kernel()
{

}
void cudamain() 
{	
	kernel<<<1,1>>>();
	printf("Hola...%i\n",1);
	return;
}


__global__ void erosion16CudaKernel(unsigned short *imgOriginalCuda,unsigned short *imgTempCuda, int mask, int rows,int cols, int step)
{
	int r, c;
    bool dark;
	r = blockIdx.x;
	c = threadIdx.x;
    dark = false;
    //Get mask
    for (int tempi=r-mask; tempi<(r+mask);tempi++){
        if((tempi>0) && (tempi <rows)){
            for (int tempj=c-mask; tempj<(c+mask);tempj++){
              if((tempj>0) && (tempj <cols)){
                //If found a drak pixel enable fla
                if( imgOriginalCuda[(tempi * step) + (tempj * 1) ]== 0){
                  dark=true;
                }
              }else{
                dark=true;
              }
            }
        }else{
            dark=true;
        }
    }
        //If drak pixel eflag is nable then place dark pixel otherwise copy original image pixel
    if( dark){
        imgTempCuda[(r * step) + (c * 1) ] = 0;
    }
    else{
        imgTempCuda[(r * step) + (c * 1) ] = imgOriginalCuda[(r * step) + (c * 1) ];
    }
}
/*
__global__ void dilation16CudaKernel(cv::Mat &img,cv::Mat &imgTemp, int mask)
{
	printf("Starting...\n");

}
__global__ void edge16CudaKernel(cv::Mat &img,cv::Mat &imgEdge, int threshold16)
{
	printf("Starting...\n");

}

__global__ void erosion8CudaKernel(cv::Mat &img,cv::Mat &imgTemp, int mask)
{
	printf("Starting...\n");

}
__global__ void dilation8CudaKernel(cv::Mat &img,cv::Mat &imgTemp, int mask)
{
	printf("Starting...\n");

}


*/

void erosion16Cuda(cv::Mat &img,cv::Mat &imgTemp, int mask){
    cv::cuda::GpuMat gpuSOurce,gpuDst;
    gpuSOurce.upload(img);
    
    unsigned short *imgOriginalCuda;
    unsigned short *imgTempCuda;



    hipMalloc((void **)&imgOriginalCuda,   gpuSOurce.rows*gpuSOurce.step);
    hipMalloc((void **)&imgTempCuda,       gpuSOurce.rows*gpuSOurce.step);

    hipMemcpyAsync(imgOriginalCuda, gpuSOurce.ptr<unsigned short>(), gpuSOurce.rows*gpuSOurce.step, hipMemcpyDeviceToDevice);

	erosion16CudaKernel<<<gpuSOurce.rows, gpuSOurce.cols>>> (imgOriginalCuda, imgTempCuda,mask,gpuSOurce.rows,gpuSOurce.cols,gpuSOurce.step);

    hipMemcpy(gpuSOurce.ptr<unsigned short>(), imgOriginalCuda, gpuSOurce.rows*gpuSOurce.step, hipMemcpyDeviceToHost);

    // following code is just for testing and visualization...
    gpuSOurce.download(img);
    
}
/*
void dilation16Cuda		(cv::Mat &img,cv::Mat &imgTemp, int mask){

}
void edge16Cuda			(cv::Mat &img,cv::Mat &imgEdge, int threshold16){

}
void erosion8Cuda		(cv::Mat &img,cv::Mat &imgTemp, int mask){

}
void dilation8Cuda		(cv::Mat &img,cv::Mat &imgTemp, int mask){

}
*/